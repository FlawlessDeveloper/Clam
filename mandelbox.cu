#include "hip/hip_runtime.h"
// for ide
#ifndef __constant__
#define __constant__
#endif
#ifndef __device__
#define __device__
#endif
#ifndef __global__
#define __global__
#endif

#include "vector.h"

struct MandelboxState
{
    Vector4<float> color;
};

struct MandelboxCfg
{
    MandelboxState* scratch;
    uint2* randbuf;
    float posX;
    float posY;
    float posZ;
    float lookX;
    float lookY;
    float lookZ;
    float upX;
    float upY;
    float upZ;
    float fov;
    float focalDistance;
    float Scale;
    float FoldingLimit;
    float FixedRadius2;
    float MinRadius2;
    float DofAmount;
    float LightPosX;
    float LightPosY;
    float LightPosZ;
    int WhiteClamp;
    float LightBrightnessHue;
    float LightBrightnessSat;
    float LightBrightnessVal;
    float AmbientBrightnessHue;
    float AmbientBrightnessSat;
    float AmbientBrightnessVal;
    float ReflectBrightnessHue;
    float ReflectBrightnessSat;
    float ReflectBrightnessVal;
    int MaxIters;
    float Bailout;
    float DeMultiplier;
    int RandSeedInitSteps;
    float MaxRayDist;
    int MaxRaySteps;
    int NumRayBounces;
    float QualityFirstRay;
    float QualityRestRay;
};

__constant__ MandelboxCfg CfgArr;
#define cfg (CfgArr)

/*
namespace staticassert
{
    template<bool x>
    struct test;
    template<>
    struct test<true>
    {
    };
    template<int x, int y>
    struct eq
    {
        test<x == y> foo;
    };
};

static __device__ void Test()
{
    staticassert::eq<sizeof(MandelboxState), MandelboxStateSize> bar;
    (void)bar.foo;
}
*/

static __device__ Vector3<float> xyz(Vector4<float> val)
{
    return Vector3<float>(val.x, val.y, val.z);
}

struct Random
{
    unsigned long long seed;
    __device__ Random(uint2 *randBufferArr, int idx, bool init)
    {
        uint2 randBuffer = randBufferArr[idx];
        seed = (unsigned long long)randBuffer.x << 32 | (unsigned long long)randBuffer.y;
        seed += idx;
        if (init)
        {
            for (int i = 0;
                 (float)i / cfg.RandSeedInitSteps - 1 < Next() * cfg.RandSeedInitSteps;
                 i++)
            {
            }
        }
    }

    __device__ void Save(uint2 *randBufferArr, int idx) const
    {
        randBufferArr[idx] = make_uint2((unsigned int)(seed >> 32), (unsigned int)seed);
    }

    __device__ unsigned int MWC64X()
    {
        unsigned int c = seed >> 32, x = seed & 0xFFFFFFFF;
        seed = x * ((unsigned long long)4294883355U) + c;
        return x ^ c;
    }

    __device__ float Next()
    {
        return (float)MWC64X() / 4294967296.0f;
    }

    __device__ Vector2<float> Circle()
    {
        Vector2<float> polar = Vector2<float>(Next() * 6.28318531f, sqrtf(Next()));
        return Vector2<float>(cos(polar.x) * polar.y, sin(polar.x) * polar.y);
    }

    __device__ Vector2<float> Normal()
    {
        // Box-Muller transform
        // returns two normally-distributed independent variables
        float mul = sqrtf(-2 * log2(Next()));
        float angle = 6.28318530718f * Next();
        return mul * Vector2<float>(cos(angle), sin(angle));
    }

    __device__ Vector3<float> Sphere()
    {
        Vector2<float> temp = Normal();
        Vector3<float> result = Vector3<float>(temp.x, temp.y, Normal().x);
        if (result.length2() != 0)
        {
            return result.normalized();
        }
        else
        {
            return Vector3<float>(1, 0, 0);
        }
    }

    // __device__ Vector3<float> Hemisphere(Vector3<float> normal)
    // {
    //     Vector3<float> result = Sphere();
    //     result *= dot(result, normal) > 0 ? 1.0f : -1.0f;
    //     return result;
    // }
};

struct Ray
{
    Vector3<float> pos;
    Vector3<float> dir;
    __device__ Ray(Vector3<float> pos, Vector3<float> dir) : pos(pos), dir(dir)
    {
    }

    // http://en.wikipedia.org/wiki/Stereographic_projection
    static __device__ Vector3<float>
    RayDir(Vector3<float> forward, Vector3<float> up, Vector2<float> screenCoords, float fov)
    {
        screenCoords *= -fov;
        float len2 = screenCoords.length2();
        Vector3<float> look =
            Vector3<float>(2 * screenCoords.x, 2 * screenCoords.y, len2 - 1) / -(len2 + 1);
        Vector3<float> right = cross(forward, up);
        return look.x * right + look.y * up + look.z * forward;
    }

    __device__ Vector3<float> At(float time)
    {
        return pos + dir * time;
    }

    __device__ void Dof(float focalPlane, Random &rand)
    {
        Vector3<float> focalPosition = At(focalPlane);
        Vector3<float> xShift = cross(Vector3<float>(0, 0, 1), dir);
        Vector3<float> yShift = cross(dir, xShift);
        Vector2<float> offset = rand.Circle();
        float dofPickup = cfg.DofAmount;
        dir = (dir + offset.x * dofPickup * xShift + offset.y * dofPickup * yShift).normalized();
        pos = focalPosition - dir * focalPlane;
    }

    static __device__ Ray
    Camera(int x, int y, int screenX, int screenY, int width, int height, Random &rand)
    {
        Vector3<float> origin = Vector3<float>(cfg.posX, cfg.posY, cfg.posZ);
        Vector3<float> look = Vector3<float>(cfg.lookX, cfg.lookY, cfg.lookZ);
        Vector3<float> up = Vector3<float>(cfg.upX, cfg.upY, cfg.upZ);
        Vector2<float> screenCoords = Vector2<float>((float)(x + screenX), (float)(y + screenY));
        screenCoords += Vector2<float>(rand.Next() - 0.5f, rand.Next() - 0.5f);
        float fov = cfg.fov * 2 / (width + height);
        Vector3<float> direction = RayDir(look, up, screenCoords, fov);
        Ray result(origin, direction);
        result.Dof(cfg.focalDistance, rand);
        return result;
    }
};

static __device__ Vector3<float> HueToRGB(float hue, float saturation, float value)
{
    hue *= 3;
    float frac = fmod(hue, 1.0f);
    Vector3<float> color;
    switch ((int)hue)
    {
        case 0:
            color = Vector3<float>(1 - frac, frac, 0);
            break;
        case 1:
            color = Vector3<float>(0, 1 - frac, frac);
            break;
        case 2:
            color = Vector3<float>(frac, 0, 1 - frac);
            break;
        default:
            color = Vector3<float>(1, 1, 1);
            break;
    }
    saturation = value * (1 - saturation);
    color = color * (value - saturation) + Vector3<float>(saturation, saturation, saturation);
    return color;
}

struct Fractal
{
    static __device__ Vector4<float> Mandelbulb(Vector4<float> z, const float Power)
    {
        const float r = xyz(z).length();

        // convert to polar coordinates
        float theta = asin(z.z / r);
        float phi = atan2(z.y, z.x);
        float dr = powf(r, Power - 1.0) * Power * z.w + 1.0;

        // scale and rotate the point
        float zr = pow(r, Power);
        theta = theta * Power;
        phi = phi * Power;

        // convert back to cartesian coordinates
        Vector3<float> z3 =
            zr * Vector3<float>(cos(theta) * cos(phi), cos(theta) * sin(phi), sin(theta));
        return Vector4<float>(z3.x, z3.y, z3.z, dr);
    }

    static __device__ Vector4<float> BoxfoldD(Vector4<float> z)
    {
        Vector3<float> znew = xyz(z);
        znew = znew.clamp(-cfg.FoldingLimit, cfg.FoldingLimit) * 2.0f - znew;
        return Vector4<float>(znew.x, znew.y, znew.z, z.w);
    }

    static __device__ Vector4<float> ContBoxfoldD(Vector4<float> z)
    {
        Vector3<float> znew = xyz(z);
        Vector3<float> zsq = Vector3<float>(znew.x * znew.x, znew.y * znew.y, znew.z * znew.z);
        zsq += Vector3<float>(1, 1, 1);
        Vector3<float> res = Vector3<float>(
            znew.x / sqrtf(zsq.x), znew.y / sqrtf(zsq.y), znew.z / sqrtf(zsq.z));
        res *= sqrtf(8);
        res = znew - res;
        return Vector4<float>(res.x, res.y, res.z, z.w);
    }

    static __device__ Vector4<float> SpherefoldD(Vector4<float> z)
    {
        z *= cfg.FixedRadius2 / clamp(xyz(z).length2(), cfg.MinRadius2, cfg.FixedRadius2);
        return z;
    }

    static __device__ Vector4<float> ContSpherefoldD(Vector4<float> z)
    {
        z *= cfg.MinRadius2 / xyz(z).length2() + cfg.FixedRadius2;
        return z;
    }

    static __device__ Vector4<float> TScaleD(Vector4<float> z)
    {
        const float scale = cfg.Scale;
        const Vector4<float> mul(scale, scale, scale, fabs(scale));
        return comp_mul(z, mul);
    }

    static __device__ Vector4<float> TOffsetD(Vector4<float> z, Vector3<float> offset)
    {
        return z + Vector4<float>(offset.x, offset.y, offset.z, 1.0f);
    }

    static __device__ Vector4<float> MandelboxD(Vector4<float> z, Vector3<float> offset)
    {
        // z = ContBoxfoldD(z);
        // z = ContSpherefoldD(z);
        z = BoxfoldD(z);
        z = SpherefoldD(z);
        z = TScaleD(z);
        z = TOffsetD(z, offset);
        return z;
    }

    static __device__ float De(Vector3<float> offset)
    {
        Vector4<float> z = Vector4<float>(offset.x, offset.y, offset.z, 1.0f);
        int n = cfg.MaxIters;
        if (n < 1)
        {
            n = 1;
        }
        do
        {
            z = MandelboxD(z, offset);
        }
        while (xyz(z).length2() < cfg.Bailout && --n);
        return xyz(z).length() / z.w;
    }

    static __device__ float Cast(
        Ray ray, const float quality, const float maxDist, Vector3<float> *normal
    )
    {
        float distance;
        float totalDistance = 0.0f;
        int i = 0;
        const int maxSteps = cfg.MaxRaySteps;
        const float deMultiplier = cfg.DeMultiplier;
        do
        {
            distance = De(ray.At(totalDistance)) * deMultiplier;
            totalDistance += distance;
            if (++i == maxSteps)
            {
                *normal = Vector3<float>(1, 0, 0);
                return totalDistance;
            }
        }
        while (totalDistance < maxDist && distance * quality > totalDistance);
        Vector3<float> final = ray.At(totalDistance);
        float delta = 1e-6f; // aprox. 8.3x float epsilon
        if (distance * 0.5f > delta)
        {
            delta = distance * 0.5f;
        }
        float dnpp = De(final + Vector3<float>(-delta, delta, delta));
        float dpnp = De(final + Vector3<float>(delta, -delta, delta));
        float dppn = De(final + Vector3<float>(delta, delta, -delta));
        float dnnn = De(final + Vector3<float>(-delta, -delta, -delta));
        *normal = Vector3<float>((dppn + dpnp) - (dnpp + dnnn),
            (dppn + dnpp) - (dpnp + dnnn),
            (dpnp + dnpp) - (dppn + dnnn));
        if (normal->length2() == 0)
        {
            *normal = Vector3<float>(1, 0, 0);
        }
        *normal = normal->normalized();
        return totalDistance;
    }
};

struct Tracer
{
    static __device__ Vector3<float> AmbientBrightness()
    {
        return HueToRGB(cfg.AmbientBrightnessHue, cfg.AmbientBrightnessSat, cfg.AmbientBrightnessVal
        );
    }

    static __device__ Vector3<float> LightBrightness()
    {
        return HueToRGB(cfg.LightBrightnessHue, cfg.LightBrightnessSat, cfg.LightBrightnessVal);
    }

    static __device__ Vector3<float> ReflectBrightness()
    {
        return HueToRGB(cfg.ReflectBrightnessHue, cfg.ReflectBrightnessSat, cfg.ReflectBrightnessVal);
    }

    static __device__ Vector3<float> LightPos()
    {
        return Vector3<float>(cfg.LightPosX, cfg.LightPosY, cfg.LightPosZ);
    }

    static __device__ float
    Specular(Vector3<float> incoming, Vector3<float> outgoing, Vector3<float> normal)
    {
        const Vector3<float> half = (incoming + outgoing).normalized();
        const float dot_prod = fabsf(dot(half, normal));
        const float hardness = 128.0f;
        //const float base_reflection = 0.7f; // TODO: Make configurable.
        const float base_reflection = cfg.ReflectBrightnessVal;
        const float spec = powf(sinpif(dot_prod * 0.5f), hardness);
        return spec * (1 - base_reflection) + base_reflection;
    }

    static __device__ Vector3<float> Trace(Ray ray, int width, int height, Random &rand)
    {
        Vector3<float> rayColor(0, 0, 0);
        float reflectionColor = 1.0f;
        bool firstRay = true;
        const float maxDist = cfg.MaxRayDist;
        for (int i = 0; i < cfg.NumRayBounces; i++)
        {
            const float quality = firstRay ? cfg.QualityFirstRay *
                ((width + height) / (2 * cfg.fov)) : cfg.QualityRestRay;
            Vector3<float> normal;
            const float distance = Fractal::Cast(ray, quality, maxDist, &normal);
            if (distance > maxDist)
            {
                Vector3<float> color = firstRay
                    ? Vector3<float>(0.3, 0.3, 0.3)
                    : AmbientBrightness();
                rayColor += color * reflectionColor;
                break;
            }
            // incorporates lambertian lighting
            const Vector3<float> newDir = (rand.Sphere() + normal).normalized();
            const Vector3<float> newPos = ray.At(distance);

            // direct lighting
            // TODO: Soft shadows, shuffle lightPos a bit
            const Vector3<float> toLightVec = LightPos() - newPos;
            const float lightDist = toLightVec.length();
            const Vector3<float> toLight = toLightVec * (1 / lightDist);
            float normalDotProd = dot(normal, toLight);
            if (normalDotProd > 0)
            {
                const float dimmingFactor =
                    (normalDotProd * reflectionColor * Specular(toLight, -ray.dir, normal)) /
                        (lightDist * lightDist);
                const float distance = Fractal::Cast(
                    Ray(newPos, toLight), quality, lightDist, &normal
                );
                if (distance >= lightDist)
                {
                    rayColor += LightBrightness() * dimmingFactor;
                }
            }
            reflectionColor *= Specular(newDir, -ray.dir, normal);
            ray = Ray(newPos, newDir);
            firstRay = false;
        }
        return rayColor;
    }
};

static __device__ unsigned int PackPixel(Vector3<float> pixel)
{
    const float gamma_correct = 1 / 2.2f;
    pixel.x = powf(pixel.x, gamma_correct);
    pixel.y = powf(pixel.y, gamma_correct);
    pixel.z = powf(pixel.z, gamma_correct);
    if (cfg.WhiteClamp)
    {
        float maxVal = max(max(pixel.x, pixel.y), pixel.z);
        if (maxVal > 1)
        {
            pixel *= 1.0f / maxVal;
        }
    }
    else
    {
        pixel = pixel.clamp(0.0f, 1.0f);
    }
    pixel = comp_mul(pixel, Vector3<float>(255, 255, 255));
    return ((unsigned int)255 << 24) | ((unsigned int)(unsigned char)pixel.x << 16) |
        ((unsigned int)(unsigned char)pixel.y << 8) | ((unsigned int)(unsigned char)pixel.z);
}

// type: -1 is preview, 0 is init, 1 is continue
extern "C" __global__ void Main(
    unsigned int *__restrict__ screenPixels,
    int screenX,
    int screenY,
    int width,
    int height,
    int frame
)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int x = idx % width;
    int y = idx / width;
    if (y >= height)
    {
        return;
    }
    // flip image - in screen space, 0,0 is top-left, in 3d space, 0,0 is bottom-left
    y = height - (y + 1);
    Random rand(static_cast<uint2 *>(cfg.randbuf), idx, frame <= 0);
    Ray ray = Ray::Camera(x, y, screenX, screenY, width, height, rand);
    Vector3<float> color = Tracer::Trace(ray, width, height, rand);
    MandelboxState *scratch = &cfg.scratch[idx];
    Vector4<float> oldColor = frame > 0 ? scratch->color : Vector4<float>(0, 0, 0, 0);
    float newWeight = oldColor.w + 1;
    Vector3<float> newColor = (color + xyz(oldColor) * oldColor.w) / newWeight;
    scratch->color = Vector4<float>(newColor.x, newColor.y, newColor.z, newWeight);

    int packedColor = PackPixel(newColor);
    screenPixels[idx] = packedColor;
    rand.Save(cfg.randbuf, idx);
}
