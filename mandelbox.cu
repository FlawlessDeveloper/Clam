#include "hip/hip_runtime.h"
#include <../samples/common/inc/hip/hip_vector_types.h>
#include <float.h>
#include "mandelbox.h"

#define Gauss(a, c, w, x) ((a) * exp(-(((x) - (c)) * ((x) - (c))) / (float)(2 * (w) * (w))))

__constant__ MandelboxCfg MandelboxCfgArr[1];
#define cfg (MandelboxCfgArr[0])

// http://en.wikipedia.org/wiki/Stereographic_projection
__device__ float3 RayDir(float3 forward, float3 up, float2 screenCoords, float fov) {
    screenCoords *= -fov;
    float len2 = dot(screenCoords, screenCoords);
    float3 look = make_float3(2 * screenCoords.x, 2 * screenCoords.y, len2 - 1)
            / -(len2 + 1);

    float3 right = cross(forward, up);

    return look.x * right + look.y * up + look.z * forward;
}

__device__ float3 Rotate(float3 a, float angle)
{
    return make_float3(
        cos(angle) * a.x - sin(angle) * a.y,
        sin(angle) * a.x + cos(angle) * a.y,
        a.z
    );
}

__device__ float De(float3 offset) {
    offset = Rotate(offset, cfg.InitRotation);
    float4 z = make_float4(offset, 1.0f);
    const float FoldingLimit = cfg.FoldingLimit;
    const float MinRadius2 = cfg.MinRadius2;
    const float FixedRadius2 = cfg.FixedRadius2;
    const float Scale = cfg.Scale;
    const float DeRotation = cfg.DeRotation;
    for (int n = 0; n < MaxIters; n++) {
        float3 znew = make_float3(z.x, z.y, z.z);
        znew = clamp(znew, -FoldingLimit, FoldingLimit) * 2.0f - znew;
        z = make_float4(znew.x, znew.y, znew.z, z.w);

        float len2 = dot(znew, znew);
        if (len2 > Bailout)
            break;
        z *= FixedRadius2 / clamp(len2, MinRadius2, FixedRadius2);

        z = make_float4(Scale, Scale, Scale, fabs(Scale)) * z
                + make_float4(offset.x, offset.y, offset.z, 1.0f);

        float3 zRot = make_float3(z.x, z.y, z.z);
        zRot = Rotate(zRot, DeRotation);
        z = make_float4(zRot.x, zRot.y, zRot.z, z.w);
    }
    float3 zxyz = make_float3(z.x, z.y, z.z);
    return length(zxyz) / z.w;
}

__device__ float DeColor(float3 offset, float lightHue) {
    offset = Rotate(offset, cfg.InitRotation);
    float3 z = offset;
    int hue = 0;
    const float FoldingLimit = cfg.FoldingLimit;
    const float MinRadius2 = cfg.MinRadius2;
    const float FixedRadius2 = cfg.FixedRadius2;
    const float Scale = cfg.Scale;
    const float DeRotation = cfg.DeRotation;
    for (int n = 0; n < MaxIters && dot(z, z) < Bailout; n++) {
        float3 zold = z;
        z = clamp(z, -FoldingLimit, FoldingLimit) * 2.0f - z;
        if (dot(zold - z, zold - z) > 0.0001f)
            hue += 7;
        else
            hue += 1;

        float len2 = dot(z, z);
        if (len2 > Bailout)
            break;
        float temp = FixedRadius2 / clamp(len2, MinRadius2, FixedRadius2);
        z *= temp;
        if (len2 < MinRadius2)
            hue += 0;
        else if (len2 < FixedRadius2)
            hue += 2;
        else
            hue += 5;

        z = Scale * z + offset;

        z = Rotate(z, DeRotation);
    }
    float fullValue = lightHue - hue * cfg.HueVariance;
    fullValue *= 3.14159f;
    fullValue = cos(fullValue);
    fullValue *= fullValue;
    fullValue = pow(fullValue, cfg.ColorSharpness);
    fullValue = 1 - (1 - fullValue) * cfg.Saturation;
    return fullValue;
}

__device__ uint MWC64X(ulong *state)
{
    uint c=(*state)>>32, x=(*state)&0xFFFFFFFF;
    *state = x*((ulong)4294883355U) + c;
    return x^c;
}

__device__ float Rand(ulong* seed)
{
    return (float)MWC64X(seed) / UINT_MAX;
}

__device__ float2 RandCircle(ulong* rand)
{
    float2 polar = make_float2(Rand(rand) * 6.28318531f, sqrt(Rand(rand)));
    return make_float2(cos(polar.x) * polar.y, sin(polar.x) * polar.y);
}

// Box-Muller transform
// returns two normally-distributed independent variables
__device__ float2 RandNormal(ulong* rand)
{
    float mul = sqrt(-2 * log2(Rand(rand)));
    float angle = 6.28318530718f * Rand(rand);
    return mul * make_float2(cos(angle), sin(angle));
}

__device__ float3 RandSphere(ulong* rand)
{
    float2 normal;
    float rest;
    do
    {
        normal = RandNormal(rand);
        rest = RandNormal(rand).x;
    } while (normal.x == 0 && normal.y == 0 && rest == 0);
    return normalize(make_float3(normal.x, normal.y, rest));
}

__device__ float3 RandHemisphere(ulong* rand, float3 normal)
{
    float3 result = RandSphere(rand);
    if (dot(result, normal) < 0)
        result = -result;
    return result;
}

__device__ void ApplyDof(float3* position, float3* lookat, float focalPlane, float hue, ulong* rand)
{
    float3 focalPosition = *position + *lookat * focalPlane;
    float3 xShift = cross(make_float3(0, 0, 1), *lookat);
    float3 yShift = cross(*lookat, xShift);
    float2 offset = RandCircle(rand);
    float dofPickup = cfg.DofAmount;
    *lookat = normalize(*lookat + offset.x * dofPickup * xShift + offset.y * dofPickup * yShift);
    *position = focalPosition - *lookat * focalPlane;
}

__device__ float3 Normal(float3 pos) {
    const float delta = FLT_EPSILON * 2;
    float dppn = De(pos + make_float3(delta, delta, -delta));
    float dpnp = De(pos + make_float3(delta, -delta, delta));
    float dnpp = De(pos + make_float3(-delta, delta, delta));
    float dnnn = De(pos + make_float3(-delta, -delta, -delta));

    return normalize(make_float3(
                (dppn + dpnp) - (dnpp + dnnn),
                (dppn + dnpp) - (dpnp + dnnn),
                (dpnp + dnpp) - (dppn + dnnn)
                ));
}

__device__ float RaySphereIntersection(float3 rayOrigin, float3 rayDir,
    float3 sphereCenter, float sphereSize,
    bool canBePast)
{
    float3 omC = rayOrigin - sphereCenter;
    float lDotOmC = dot(rayDir, omC);
    float underSqrt = lDotOmC * lDotOmC - dot(omC, omC) + sphereSize * sphereSize;
    if (underSqrt < 0)
        return FLT_MAX;
    float theSqrt = sqrt(underSqrt);
    float dist = -lDotOmC - theSqrt;
    if (dist > 0)
        return dist;
    dist = -lDotOmC + theSqrt;
    if (canBePast && dist > 0)
        return dist;
    return FLT_MAX;
}

__device__ float Trace(float3 origin, float3 direction, float quality, float hue, ulong* rand,
        int* isFog, int* hitLightsource)
{
    float distance = 1.0f;
    float totalDistance = De(origin) * DeMultiplier * Rand(rand);
    const float3 lightPos = make_float3(cfg.LightPosX, cfg.LightPosY, cfg.LightPosZ);
    float sphereDist = RaySphereIntersection(origin, direction, lightPos, cfg.LightSize, false);
    float fogDist = -log2(Rand(rand)) / (float)(cfg.FogDensity /* * hue */);
    float maxRayDist = min(min((float)MaxRayDist, fogDist), sphereDist);
    for (int i = 0; i < MaxRaySteps && totalDistance < maxRayDist &&
            distance * quality > totalDistance; i++) {
        distance = De(origin + direction * totalDistance) * DeMultiplier;
        totalDistance += distance;
    }
    if (totalDistance > sphereDist)
        *hitLightsource = 1;
    else
        *hitLightsource = 0;
    if (totalDistance > fogDist)
    {
        *isFog = 1;
        totalDistance = fogDist;
    }
    else if (totalDistance > MaxRayDist)
        *isFog = 1;
    else
        *isFog = 0;
    return totalDistance;
}

__device__ float SimpleTrace(float3 origin, float3 direction, float quality)
{
    float distance = 1.0f;
    float totalDistance = 0.0f;
    const float3 lightPos = make_float3(cfg.LightPosX, cfg.LightPosY, cfg.LightPosZ);
    float sphereDist = RaySphereIntersection(origin, direction, lightPos, cfg.LightSize, false);
    float maxRayDist = min((float)MaxRayDist, sphereDist);
    int i;
    for (i = 0; i < MaxRaySteps && totalDistance < maxRayDist &&
            distance * quality > totalDistance; i++)
    {
        distance = De(origin + direction * totalDistance) * DeMultiplier;
        totalDistance += distance;
    }
    return (float)i / MaxRaySteps;
}

__device__ bool Reaches(float3 initial, float3 final)
{
    float3 direction = final - initial;
    float lenDir = length(direction);
    direction /= lenDir;
    float totalDistance = 0;
    float distance = FLT_MAX;
    float threshHold = fabs(De(final)) * (DeMultiplier * 0.5f);
    for (int i = 0; i < MaxRaySteps && totalDistance < MaxRayDist &&
                distance > threshHold; i++) {
        distance = De(initial + direction * totalDistance) * DeMultiplier;
        if (i == 0 && fabs(distance * 0.5f) < threshHold)
            threshHold = fabs(distance * 0.5f);
        totalDistance += distance;
        if (totalDistance > lenDir)
            return true;
    }
    return false;
}

__device__ float LightBrightness(float hue)
{
    return Gauss(cfg.LightBrightnessAmount, cfg.LightBrightnessCenter, cfg.LightBrightnessWidth, hue);
}

__device__ float AmbientBrightness(float hue)
{
    return Gauss(cfg.AmbientBrightnessAmount, cfg.AmbientBrightnessCenter, cfg.AmbientBrightnessWidth, hue);
}

__device__ float DirectLighting(float3 rayPos, float hue, ulong* rand, float3* lightDir)
{
    const float3 lightPos = make_float3(cfg.LightPosX, cfg.LightPosY, cfg.LightPosZ);
    float3 lightToRay = normalize(rayPos - lightPos);
    float3 movedLightPos = lightPos + cfg.LightSize * RandHemisphere(rand, lightToRay);
    *lightDir = normalize(movedLightPos - rayPos);
    if (Reaches(rayPos, movedLightPos))
    {
        float div = dot(rayPos - movedLightPos, rayPos - movedLightPos);
        return LightBrightness(hue) / div;
    }
    return 0.0f;
}

__device__ float BRDF(float3 normal, float3 incoming, float3 outgoing)
{
    float3 halfV = normalize(incoming + outgoing);
    float angle = acos(dot(normal, halfV));
    return 1 + Gauss(cfg.SpecularHighlightAmount, 0, cfg.SpecularHighlightSize, angle);
}


__device__ float RenderingEquation(float3 rayPos, float3 rayDir, float qualityMul, float hue, ulong* rand)
{
    float total = 0;
    float color = 1;
    int isFog;
    for (int i = 0; i < NumRayBounces; i++)
    {
        bool isQuality = i == 0;
        float quality = isQuality?QualityFirstRay*qualityMul:QualityRestRay;
        int hitLightsource;
        float distance = Trace(rayPos, rayDir, quality, hue, rand, &isFog, &hitLightsource);
        if (hitLightsource)
        {
            if (i == 0)
            {
                total += color * LightBrightness(hue);
            }
            break;
        }
        if (distance > MaxRayDist)
        {
            isFog = 1;
            break;
        }

        float3 newRayPos = rayPos + rayDir * distance;
        float3 newRayDir;

        float3 normal;
        if (isFog)
        {
            newRayDir = RandSphere(rand);
            //color *= FogColor(hue);
        }
        else
        {
            normal = Normal(newRayPos);
            newRayDir = RandHemisphere(rand, normal);
            color *= DeColor(newRayPos, hue) * cfg.Reflectivity;
        }

        float3 lightingRayDir;
        float direct = DirectLighting(newRayPos, hue, rand, &lightingRayDir);

        if (!isFog)
        {
            color *= BRDF(normal, newRayDir, -rayDir) * dot(normal, newRayDir);
            direct *= BRDF(normal, lightingRayDir, -rayDir) * dot(normal, lightingRayDir);
        }
        total += color * direct;

        rayPos = newRayPos;
        rayDir = newRayDir;

        if (isFog)
        {
            break;
        }
    }
    if (isFog)
    {
        total += color * AmbientBrightness(hue);
    }
    return total;
}

__device__ float3 HueToRGB(float hue, float value)
{
    hue *= 4;
    float frac = fmod(hue, 1.0f);
    float3 color;
    switch ((int)hue)
    {
        case 0:
            color = make_float3(frac, 0, 0);
            break;
        case 1:
            color = make_float3(1 - frac, frac, 0);
            break;
        case 2:
            color = make_float3(0, 1 - frac, frac);
            break;
        case 3:
            color = make_float3(0, 0, 1 - frac);
            break;
        default:
            color = make_float3(value);
            break;
    }
    color.x = sqrtf(color.x);
    color.y = sqrtf(color.y);
    color.z = sqrtf(color.z);
    color *= value;
    return color;
}

__device__ uint PackPixel(float4 pixel)
{
    if (cfg.WhiteClamp)
    {
        float maxVal = max(max(pixel.x, pixel.y), pixel.z);
        if (maxVal > 1)
            pixel /= maxVal;
    }
    pixel = clamp(pixel, 0.0f, 1.0f) * 255;
    return (255 << 24) | ((int)pixel.x << 16) | ((int)pixel.y << 8) | ((int)pixel.z);
}

extern "C" __global__ void kern(
        uint* __restrict__ screenPixels,
        float4* __restrict__ screen,
        uint2* __restrict__ rngBuffer,
        int screenX, int screenY, int width, int height,
        float posX, float posY, float posZ,
        float lookX, float lookY, float lookZ,
        float upX, float upY, float upZ,
        float fov, float focalDistance, float frame)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    float3 pos = make_float3(posX, posY, posZ);
    float3 look = make_float3(lookX, lookY, lookZ);
    float3 up = make_float3(upX, upY, upZ);

    ulong rand;
    uint2 randBuffer = rngBuffer[y * width + x];
    rand = (ulong)randBuffer.x << 32 | (ulong)randBuffer.y;
    rand += y * width + x;
    if (frame == 0)
    {
        for (int i = 0; (float)i / RandSeedInitSteps - 1 < Rand(&rand) * RandSeedInitSteps; i++)
        {
        }
    }

    float hue = Rand(&rand);

    float2 screenCoords = make_float2((float)(x + screenX), (float)(y + screenY));
    fov *= exp((hue - 0.5f) * cfg.FovAbberation);
    float3 rayDir = RayDir(look, up, screenCoords, fov);
    ApplyDof(&pos, &rayDir, focalDistance, hue, &rand);
    int screenIndex = y * width + x;

    float4 final;
    if (frame == 0)
    {
        float dist = SimpleTrace(pos, rayDir, 1 / fov);
        dist = sqrt(dist);
        screen[screenIndex] = final = make_float4(dist);
    }
    else
    {
        frame -= 1;

        const float weight = 1;
        float intensity = RenderingEquation(pos, rayDir, 1 / fov, hue, &rand);
        if (!cfg.BrightThresh)
        {
            intensity = fmin(intensity, cfg.BrightThresh);
        }
        float3 color = HueToRGB(hue, intensity) + make_float3(cfg.ColorBiasR, cfg.ColorBiasG, cfg.ColorBiasB);

        float4 old = screen[screenIndex];
        float3 oldxyz = make_float3(old.x, old.y, old.z);
        float3 diff = oldxyz - color;
        if (!isnan(color.x) && !isnan(color.y) && !isnan(color.z) && !isnan(weight))
        {
            if (frame != 0 && old.w + weight > 0)
                final = make_float4((color * weight + oldxyz * old.w) / (old.w + weight), old.w + weight);
            else
                final = make_float4(color, weight);
        }
        else
        {
            if (frame != 0)
                final = old;
            else
                final = make_float4(0);
        }
        screen[screenIndex] = final;
    }
    screenPixels[screenIndex] = PackPixel(final);
    rngBuffer[screenIndex] = make_uint2((uint)(rand >> 32), (uint)rand);
}
